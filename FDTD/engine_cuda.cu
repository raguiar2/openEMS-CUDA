#include "hip/hip_runtime.h"
#include "engine_cuda.h"

#include "tools/array_ops.h"

#include <hip/hip_cooperative_groups.h>

//! \brief construct an Engine instance
//! it's the responsibility of the caller to free the returned pointer
Engine_CUDA* Engine_CUDA::New(const Operator_CUDA* op, unsigned int cuda_device_number)
{
	cout << "Create FDTD engine (CUDA)" << endl;
	Engine_CUDA* e = new Engine_CUDA(op);
	e->setCUDAdevice(cuda_device_number);
	e->Init();
	return e;
}

Engine_CUDA::Engine_CUDA(const Operator_CUDA* op) : Engine::Engine(op)
{
	m_cuda_device_number = 0;
	m_supports_coop_launch = 0;
	m_gridDim = {0};
	m_blockDim = {0};
	Op = op;
	m_type = CUDA;
}

void Engine_CUDA::setCUDAdevice(unsigned int cuda_device_number) {
	m_cuda_device_number = cuda_device_number;
}

void Engine_CUDA::Init()
{
	int nDevices;
	hipGetDeviceCount(&nDevices);
  	if(nDevices <= 0)
		throw std::runtime_error("No CUDA devices found");
	if(m_cuda_device_number >= nDevices)
		throw std::runtime_error("CUDA device number out of range");
	m_cuda_device_number = m_cuda_device_number;
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, m_cuda_device_number);
    cout << "  Running on device: " << prop.name << endl;

	hipSetDevice(m_cuda_device_number);
	hipDeviceGetAttribute(&m_supports_coop_launch, hipDeviceAttributeCooperativeLaunch, m_cuda_device_number);

	m_blockDim = {10, 10, 10};
	m_gridDim = {
		(unsigned int)ceilf(numLines[0]/m_blockDim.x),
		(unsigned int)ceilf(numLines[1]/m_blockDim.y),
		(unsigned int)ceilf(numLines[2]/m_blockDim.z)
	};

	cout << "numLines: " << numLines[0] << ", " << numLines[1] << ", " << numLines[2] << endl;
	cout << "grid dim: " << m_gridDim.x << ", " << m_gridDim.y << ", " << m_gridDim.z << endl;

	numTS = 0;
	volt = Create3DArray_CUDA<CUDA_VECTOR>(numLines);
	curr = Create3DArray_CUDA<CUDA_VECTOR>(numLines);

	InitExtensions();
	SortExtensionByPriority();
}

void Engine_CUDA::Reset()
{
	Delete3DArray_CUDA(volt,numLines);
	volt=NULL;
	Delete3DArray_CUDA(curr,numLines);
	curr=NULL;

	ClearExtensions();
}

__device__ void UpdateVoltages(CUDA_VECTOR ***volt, CUDA_VECTOR ***curr, CUDA_VECTOR ***opvi, CUDA_VECTOR ***opvv, unsigned int x, unsigned int y, unsigned int z)
{
	CUDA_VECTOR v = volt[x][y][z];
	CUDA_VECTOR i = curr[x][y][z];
	CUDA_VECTOR ix = curr[x-(x!=0)][y][z];
	CUDA_VECTOR iy = curr[x][y-(y!=0)][z];
	CUDA_VECTOR iz = curr[x][y][z-(z!=0)];
	CUDA_VECTOR vi = opvi[x][y][z];
	CUDA_VECTOR vv = opvv[x][y][z];

	v.x = v.x * vv.x + vi.x * (i.z - iy.z - i.y + iz.y);
	v.y = v.y * vv.y + vi.y * (i.x - iz.x - i.z + ix.z);
	v.z = v.y * vv.y + vi.z * (i.y - ix.y - i.x + iy.x);

	volt[x][y][z] = v;
}

__device__ void UpdateCurrents(CUDA_VECTOR ***volt, CUDA_VECTOR ***curr, CUDA_VECTOR ***opiv, CUDA_VECTOR ***opii, unsigned int x, unsigned int y, unsigned int z)
{
	CUDA_VECTOR i = curr[x][y][z];
	CUDA_VECTOR v = volt[x][y][z];
	CUDA_VECTOR vx = volt[x+1][y][z];
	CUDA_VECTOR vy = volt[x][y+1][z];
	CUDA_VECTOR vz = volt[z][y][z+1];
	CUDA_VECTOR iv = opiv[x][y][z];
	CUDA_VECTOR ii = opii[x][y][z];

	i.x = i.x * ii.x + iv.x * (v.z - vy.z - v.y + vz.y);
	i.y = i.y * ii.y + iv.y * (v.x - vz.x - v.z + vx.z);
	i.z = i.z * ii.z + iv.z * (v.y - vx.y - v.x + vy.x);

	curr[x][y][z] = i;
}

/*__global__ void ManyTS(Engine_CUDA *instance, unsigned int startTS, unsigned int iterTS, unsigned int numLines[])
{
	const unsigned int pos[] = {
		blockDim.x * blockIdx.x + threadIdx.x,
		blockDim.y * blockIdx.y + threadIdx.y,
		blockDim.z * blockIdx.z + threadIdx.z
	};

	if(pos[0] >= numLines[0] || pos[1] >= numLines[1] || pos[2] >= numLines[2]) return;

	const unsigned int endTS = startTS + iterTS;
	bool runUpdateCurrents = (pos[0] < numLines[0]-1 && pos[1] < numLines[1]-1 && pos[2] < numLines[2]-1);

	cooperative_groups::grid_group grid = cooperative_groups::this_grid();

	for (unsigned int ts=startTS; ts<endTS; ++ts)
	{
		//voltage updates with extensions
		//DoPreVoltageUpdates();
		UpdateVoltages(instance, pos[0], pos[1], pos[2]);
		//DoPostVoltageUpdates();
		//Apply2Voltages();

		grid.sync();

		//current updates with extensions
		//DoPreCurrentUpdates();
		if(runUpdateCurrents) UpdateCurrents(instance, pos[0], pos[1], pos[2]);
		//DoPostCurrentUpdates();
		//Apply2Current();

		grid.sync();
	}
}*/

__global__ void VoltageKernel(CUDA_VECTOR ***volt, CUDA_VECTOR ***curr, CUDA_VECTOR ***opvi, CUDA_VECTOR ***opvv, unsigned int numLinesX, unsigned int numLinesY, unsigned int numLinesZ) {
	const unsigned int pos[] = {
		blockDim.x * blockIdx.x + threadIdx.x,
		blockDim.y * blockIdx.y + threadIdx.y,
		blockDim.z * blockIdx.z + threadIdx.z
	};

	if(pos[0] >= numLinesX || pos[1] >= numLinesY || pos[2] >= numLinesZ) return;

	UpdateVoltages(volt, curr, opvi, opvv, pos[0], pos[1], pos[2]);
}

__global__ void CurrentKernel(CUDA_VECTOR ***volt, CUDA_VECTOR ***curr, CUDA_VECTOR ***opvi, CUDA_VECTOR ***opvv, unsigned int numLinesX, unsigned int numLinesY, unsigned int numLinesZ) {
	const unsigned int pos[] = {
		blockDim.x * blockIdx.x + threadIdx.x,
		blockDim.y * blockIdx.y + threadIdx.y,
		blockDim.z * blockIdx.z + threadIdx.z
	};

	if(pos[0] >= numLinesX-1 || pos[1] >= numLinesY-1 || pos[2] >= numLinesZ-1) return;

	UpdateCurrents(volt, curr, opvi, opvv, pos[0], pos[1], pos[2]);
}

bool Engine_CUDA::IterateTS(unsigned int iterTS)
{
	/*
	Engine_CUDA *instance = this;
	void *args[] = {
		&instance,
		&numTS,
		&iterTS,
		&numLines
	};
	hipError_t err = hipLaunchCooperativeKernel((void*)&ManyTS, m_gridDim, m_blockDim, args);
	if(err) throw std::runtime_error("CUDA kernel launch failure: " + std::string(hipGetErrorString(err)));
	numTS += iterTS;
	*/

	unsigned int endTS = numTS + iterTS;
	for(; numTS < endTS; numTS++)
	{
		VoltageKernel<<<m_gridDim, m_blockDim>>>(volt, curr, Op->vi, Op->vv, numLines[0], numLines[1], numLines[2]);
		hipError_t err = hipGetLastError();
		if(err) throw std::runtime_error("CUDA kernel launch failure: " + std::string(hipGetErrorString(err)));
		CurrentKernel<<<m_gridDim, m_blockDim>>>(volt, curr, Op->iv, Op->ii, numLines[0], numLines[1], numLines[2]);
		err = hipGetLastError();
		if(err) throw std::runtime_error("CUDA kernel launch failure: " + std::string(hipGetErrorString(err)));
	}
	hipDeviceSynchronize();
	return true;
}
